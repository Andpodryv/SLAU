#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <locale>
#include <Windows.h>
#define N 5
#define Block_size 1


__global__ void Str_SLAU(double* A,double *B)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x ;
    double d = 0;

   if (k<N){
        for (int j = k + 1; j < N; j++) {
            d = (double)A[j*N + k ] / A[k*N +  k];
            for (int i = k; i < N; i++)
                A[j*N + i] -= d * A[k*N + i];
            B[j] -= d * B[k];
        }
    }
}
__global__ void rev_SLAU(double* A, double* B, double* X)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    double d = 0;
    if (k < N) {
        for (int j = k + 1; j < N; j++) {
            d += A[k*N + j] * X[j];
        }
        X[k] = (B[k] - d) / A[k*N + k];
    }
}
int main() {
    setlocale(LC_ALL, "Russian");
    int i, k, j, p = 0;
    double d;
    double* A_d;
    double* B_d;
    double* X_d;
    double* HostA;
    double* HostB;
    double* HostX1;
    double* HostX2;
    HostA = (double*)calloc(N * N, sizeof(double*));
    HostB = (double*)calloc(N, sizeof(double));
    HostX1 = (double*)calloc(N, sizeof(double));
    HostX2 = (double*)calloc(N, sizeof(double));
    srand(time(NULL));
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            HostA[i*N + j ] = double(rand() - rand()) / 2020;
        }
        HostB[i] = double(rand() - rand()) / 2020;
    }
    hipMalloc((void**)&A_d, N * N * sizeof(double));
    hipMalloc((void**)&B_d, N * sizeof(double));
    hipMalloc((void**)&X_d, N * sizeof(double));
    hipMemcpy(A_d, HostA, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_d, HostB, N * sizeof(double), hipMemcpyHostToDevice);
    dim3 threads(Block_size, Block_size);
    dim3 blocks(N / threads.x, N / threads.y);
    Str_SLAU<<< 1, N*N >>>(A_d, B_d);
    hipDeviceSynchronize();/
    hipMemcpy(HostX2, X_d, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(HostA, A_d, N*N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(HostB, B_d, N * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("|");
        for (int j = 0; j < N; j++)
            printf(" %f   ", HostA[i * N + j]);
        printf(" %f ", HostB[i]);
        printf("| \n");
    }
    for (i = 0; i < N; i++)
        printf("X%d = %f  // %f \n", i, HostX1[i], HostX2[i]);
}
